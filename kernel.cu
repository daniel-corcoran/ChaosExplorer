#include "hip/hip_runtime.h"
//---CUDA BUILD---//
//DANIEL CORCORAN IS A PROGRAMMING GOD


#include <iostream>
#include <SFML/Graphics.hpp>
#include <complex>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_complex.h>
#include <chrono>
#include <thread>
#include <cmath>
//#include <boost/multiprecision/double.hpp>

#define benchmark false

//#include <boost/multiprecision/double.hpp>
#define zoom_multiplier 4
//typedef __double double_t;
//using namespace boost::multiprecision;
class settings {
	double xMax, xMin, cMax, cMin;	//S is -2
	int precision;
	int zoom_incremental = 5;
	double resolution_multiplier; //Reccomended is 20
	double penSize, autozoom_x, autozoom_c;
	bool autozoom;
public:
	settings() {
		if (benchmark==false) {
			xMax = 2;
			xMin = -2;
			cMax = 2;
			cMin = -2;
			std::cout << "Precision? Recommended: 30\n";
			std::cin >> precision;
			std::cout << "\nPixel size? Recommended: 1\n";
			std::cin >> resolution_multiplier;



			std::cout << "zoom resolution incremental: (recommended 2-10)\n";
			std::cin >> zoom_incremental;

			std::cout << "\nAutozoom? (true, false)\n";
			std::cin >> autozoom;
			if (autozoom == true) {
				autozoom_x = -0.7336438924199521;
				autozoom_c = 0.2455211406714035;
			}
		}
		else {
			xMax = 2;
			xMin = -2;
			cMax = 2;
			cMin = -2;
			precision = 100;
			zoom_incremental = 9;
			resolution_multiplier = 1;
			autozoom_x = -0.7336438924199521;
			autozoom_c = 0.2455211406714035;


		}


	}
	bool get_autozoom() {
		return autozoom;
	}
	int get_zoomIncremental(){
		return zoom_incremental; }
	double getAZ_x() {
		return autozoom_x; 
	}
	double getAZ_y() {
		return autozoom_c;
	}
	double getMin_x()
	{
		return xMin;
	}
	double getMin_c() {
		return cMin;
	}
	double getMax_x() {
		return xMax;
	}
	double getMax_c() {
		return cMax;
	}
	double XstepSize() {
		//Screen is 1920 pixels wide. We have a image of width (100 * (xmax - xmin)), so our pixels 
		return  ((xMax - xMin) / 1920) * resolution_multiplier;
	}
	double YstepSize() {
		return ((cMax - cMin) / 1080) * resolution_multiplier;
	}
	int getPrecision() {
		return precision;
	}
	
	//void zoom(double x, double y, sf::View &myView) {
	//	myView.setCenter(sf::Vector2f(x, y));
	//	myView.setSize(myView.getSize().x / 4, myView.getSize().y / 4);
	//	xMin = myView.getCenter().x - (myView.getSize().x / 4);
	//	xMax = myView.getCenter().x + (myView.getSize().x / 4);
	//	cMin = myView.getCenter().y - (myView.getSize().y / 4);
	//	cMax = myView.getCenter().y + (myView.getSize().y / 4);
	//
//	}
	void setPrecision(int newPrecision) {
		precision = newPrecision;
	}
	int getRes() { return resolution_multiplier; }
	void zoom( double x, double y) {
		double centerX = x;
		double centerC = y;
		double sizeX = (xMax - xMin);
		double sizeC = (cMax - cMin);
		xMin = centerX - (sizeX / zoom_multiplier);
		xMax = centerX + (sizeX / zoom_multiplier);
		cMin = centerC - (sizeC / zoom_multiplier);
		cMax = centerC + (sizeC / zoom_multiplier);
	}


};
__global__
void process(int n, int iterations, double Minx, double Miny,  sf::Uint8 *pixels, double x_step, double y_step, int displayX, int displayY) {
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if (id < n) {
		float i;
		int x_index = id % displayX;
		int y_index = ((id - x_index) / displayX);
		//convert x_index to coord_pos
		double c_r = Minx + (x_index * x_step);
		double c_i = Miny + (y_index * y_step);

		double zr = 0; double zi = 0; double zrsqr = 0; double zisqr = 0;

		for (i = 0; i < iterations; i+=2) {
			if (zrsqr + zisqr > 4) {
				break;
			}
			zi = zr * zi;
			zi += zi;
			zi += c_i;
			zr = zrsqr - zisqr + c_r;
			zrsqr = zr * zr;
			zisqr = zi * zi;
			zi = zr * zi;
			zi += zi;
			zi += c_i;
			zr = zrsqr - zisqr + c_r;
			zrsqr = zr * zr;
			zisqr = zi * zi;
			

		

		}
		if ( i >= iterations ) {
			pixels[id * 4] = 30;
			pixels[4 * id + 1] = 10;
			pixels[4 * id + 2] = 60;
			pixels[4 * id + 3] = 255;
		}else {
			
			
			pixels[id * 4] = 255 * (i / iterations);
			pixels[4 * id + 1] = 100 * (i / iterations);
			pixels[4 * id + 2] = 255;
			pixels[4 * id + 3] = 255;
		}
	}
}

int main()
{

	settings instance;



	sf::RenderWindow window(sf::VideoMode(1920, 1080), "Chaos Explorer");

	int displayX = ceil(1920 / instance.getRes());
	int displayY = ceil(1080 / instance.getRes());
	sf::Uint8 *pixels = new sf::Uint8[displayX * displayY * 4];
	sf::Sprite sprite;
	int total_size = displayX * displayY; //How many elements in this array?
	std::cout << "arrays purged succesfully";

	hipMallocManaged(&pixels, 4*total_size * sizeof(sf::Uint8));
	sf::Texture texture;
	if (benchmark == true) {
		//Run simulation (zoom to pt) ten times and time the duration between it, return time in milliseconds
		sf::Image image;
			while (window.isOpen()) {

				//std::cout << "total length: " << total_size << std::endl;


				{							
					auto start = std::chrono::high_resolution_clock::now(); //Begin the benchmark


					for (int i = 0; i < 10; i++) { //Zoom ten times
						
				
						int iterations = instance.getPrecision();
						double x_step = instance.XstepSize();
						double y_step = instance.YstepSize();
						
						process << <total_size, 1 >> > (total_size, iterations, instance.getMin_x(), instance.getMin_c(), pixels, x_step, y_step, displayX, displayY);
						instance.zoom(instance.getAZ_x(), instance.getAZ_y());
						instance.setPrecision(instance.getPrecision() + instance.get_zoomIncremental());
						hipDeviceSynchronize();
						
					
						image.create(displayX, displayY, pixels);
						texture.loadFromImage(image);
						sprite.setTexture(texture);
						window.draw(sprite);
						window.display();
						
						
					}
					


					auto stop = std::chrono::high_resolution_clock::now();
					auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

					std::cout << "Score: " << duration.count() << " seconds" << std::endl;

					return 0;
				}


			}

		


	}
	else {
		while (window.isOpen()) {

			std::cout << "total length: " << total_size << std::endl;

		

			{
				//			__double myfloat;
				double Minx = instance.getMin_x();
				double Miny = instance.getMin_c();
				int iterations = instance.getPrecision();
				double x_step = instance.XstepSize();
				double y_step = instance.YstepSize();
				std::cout << "--Process Initiated--\nBetween X: " << Minx << ", " << instance.getMax_x() << "\nC: " << Miny << ", " << instance.getMax_c() << std::endl;
				std::cout << "Processing data points: sending elements to CUDA\n";

				auto start = std::chrono::high_resolution_clock::now();


				process << <total_size, 1 >> > (total_size, iterations, Minx, Miny,pixels, x_step, y_step, displayX, displayY);
				hipDeviceSynchronize();
				auto stop = std::chrono::high_resolution_clock::now();
				auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
				std::cout << "Iterative processing time (GPU): " << duration.count() / pow(10, 6) << " seconds" << std::endl;
				start = std::chrono::high_resolution_clock::now();
				std::cout << "creating image from data array...";
				sf::Image image;
				image.create(displayX, displayY, pixels);
				texture.loadFromImage(image);
				sprite.setTexture(texture);
				window.draw(sprite);
				window.display();
				stop = std::chrono::high_resolution_clock::now();
				duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);

				std::cout << "Image processing time (CPU): " << duration.count() / pow(10, 6) << " seconds" << std::endl;






				//std::cout << "system 1: \n";


				sf::Event event;


				std::cout << "render complete" << std::endl;
				bool wait = false;

				if (instance.get_autozoom() == false) {
					wait = true;
				}
				else {
					//autozoom routine;
					instance.zoom(instance.getAZ_x(), instance.getAZ_y());
					instance.setPrecision(instance.getPrecision() + 9);

				}
				while (wait) {

					while (window.pollEvent(event))
					{
						if (event.type == sf::Event::Closed) {
							//free(fz);
							//free(pixels);
							window.close();
							return 0;
						}
						if (event.type == sf::Keyboard::Escape) {
							// free(fz);
							free(pixels);
							window.close();
							return 0;
						}
						if (sf::Mouse::isButtonPressed(sf::Mouse::Left)) {
							sf::Vector2i mousepos = sf::Mouse::getPosition(window);

							//XMin + (pixels * stepx), XMin + (pixels * stepx)
							//sf::Vector2f plotpos(instance.getMin_x() + (mousepos.x * instance.XstepSize()), instance.getMin_c() + (mousepos.y * instance.YstepSize()));
							instance.zoom(instance.getMin_x() + (mousepos.x * instance.XstepSize()), instance.getMin_c() + (mousepos.y * instance.YstepSize()));

							//instance.zoom(plotpos.x, plotpos.y);
							//camera.setSize(camera.getSize().x / 2, camera.getSize().y / 2);
							instance.setPrecision(instance.getPrecision() + instance.get_zoomIncremental());

							wait = false;


						}
					}

				}




			}


		}

	}







	
	return 0;
}